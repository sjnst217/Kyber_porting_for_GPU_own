#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kem_1.cuh"
#include "type_1.cuh"
#include "fips202_1.cuh"
#include <ctype.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#define hash_h(OUT, IN, INBYTES) sha3_256(OUT, IN, INBYTES)
#define hash_g(OUT, IN, INBYTES) sha3_512(OUT, IN, INBYTES)
#define gen_a(A,B)  PQCLEAN_KYBER512_CLEAN_gen_matrix(A,B,0)
#define gen_at(A,B) PQCLEAN_KYBER512_CLEAN_gen_matrix(A,B,1)

#define xof_absorb(STATE, SEED, X, Y) PQCLEAN_KYBER512_CLEAN_kyber_shake128_absorb(STATE, SEED, X, Y)
#define xof_squeezeblocks(OUT, OUTBLOCKS, STATE) shake128_squeezeblocks(OUT, OUTBLOCKS, STATE)
#define xof_ctx_release(STATE) shake128_ctx_release(STATE)
#define prf(OUT, OUTBYTES, KEY, NONCE) PQCLEAN_KYBER512_CLEAN_kyber_shake256_prf(OUT, OUTBYTES, KEY, NONCE)
#define kdf(OUT, IN, INBYTES) shake256(OUT, KYBER_SSBYTES, IN, INBYTES)

#define GEN_MATRIX_NBLOCKS ((12*KYBER_N/8*(1 << 12)/KYBER_Q + XOF_BLOCKBYTES)/XOF_BLOCKBYTES)

#define XOF_BLOCKBYTES SHAKE128_RATE

__device__ static void randombytes_win32_randombytes(uint8_t* buf, const size_t n) {
    for (int i = 0; i < 32; i++) {
        *buf = i;
        buf++;
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_frommsg(poly* r, const uint8_t msg[KYBER_INDCPA_MSGBYTES]) {
    size_t i, j;
    int16_t mask;

    for (i = 0; i < KYBER_N / 8; i++) {
        for (j = 0; j < 8; j++) {
            mask = -(int16_t)((msg[i] >> j) & 1);
            r->coeffs[8 * i + j] = mask & ((KYBER_Q + 1) / 2);
        }
    }
}

__device__ static const uint64_t KeccakF_RoundConstants[NROUNDS] = {
    0x0000000000000001ULL, 0x0000000000008082ULL,
    0x800000000000808aULL, 0x8000000080008000ULL,
    0x000000000000808bULL, 0x0000000080000001ULL,
    0x8000000080008081ULL, 0x8000000000008009ULL,
    0x000000000000008aULL, 0x0000000000000088ULL,
    0x0000000080008009ULL, 0x000000008000000aULL,
    0x000000008000808bULL, 0x800000000000008bULL,
    0x8000000000008089ULL, 0x8000000000008003ULL,
    0x8000000000008002ULL, 0x8000000000000080ULL,
    0x000000000000800aULL, 0x800000008000000aULL,
    0x8000000080008081ULL, 0x8000000000008080ULL,
    0x0000000080000001ULL, 0x8000000080008008ULL
};
__device__ const int16_t PQCLEAN_KYBER512_CLEAN_zetas[128] = {
    -1044,  -758,  -359, -1517,  1493,  1422,   287,   202,
        -171,   622,  1577,   182,   962, -1202, -1474,  1468,
        573, -1325,   264,   383,  -829,  1458, -1602,  -130,
        -681,  1017,   732,   608, -1542,   411,  -205, -1571,
        1223,   652,  -552,  1015, -1293,  1491,  -282, -1544,
        516,    -8,  -320,  -666, -1618, -1162,   126,  1469,
        -853,   -90,  -271,   830,   107, -1421,  -247,  -951,
        -398,   961, -1508,  -725,   448, -1065,   677, -1275,
        -1103,   430,   555,   843, -1251,   871,  1550,   105,
        422,   587,   177,  -235,  -291,  -460,  1574,  1653,
        -246,   778,  1159,  -147,  -777,  1483,  -602,  1119,
        -1590,   644,  -872,   349,   418,   329,  -156,   -75,
        817,  1097,   603,   610,  1322, -1285, -1465,   384,
        -1215,  -136,  1218, -1335,  -874,   220, -1187, -1659,
        -1185, -1530, -1278,   794, -1510,  -854,  -870,   478,
        -108,  -308,   996,   991,   958, -1460,  1522,  1628
};

__device__ static void store64(uint8_t* x, uint64_t u) {
    for (size_t i = 0; i < 8; ++i) {
        x[i] = (uint8_t)(u >> 8 * i);
    }
}
__device__ static uint64_t load64(const uint8_t* x) {
    uint64_t r = 0;
    for (size_t i = 0; i < 8; ++i) {
        r |= (uint64_t)x[i] << 8 * i;
    }

    return r;
}
__device__ static uint32_t load24_littleendian(const uint8_t x[3])
{
    uint32_t r;
    r = (uint32_t)x[0];
    r |= (uint32_t)x[1] << 8;
    r |= (uint32_t)x[2] << 16;
    return r;
}
__device__ static uint32_t load32_littleendian(const uint8_t x[4]) {
    uint32_t r;
    r = (uint32_t)x[0];
    r |= (uint32_t)x[1] << 8;
    r |= (uint32_t)x[2] << 16;
    r |= (uint32_t)x[3] << 24;
    return r;
}

__device__ void PQCLEAN_KYBER512_CLEAN_poly_tobytes(uint8_t r[KYBER_POLYBYTES], const poly* a)
{
    size_t i;
    uint16_t t0, t1;

    for (i = 0; i < KYBER_N / 2; i++) {
        // map to positive standard representatives
        t0 = a->coeffs[2 * i];
        t0 += ((int16_t)t0 >> 15) & KYBER_Q;
        t1 = a->coeffs[2 * i + 1];
        t1 += ((int16_t)t1 >> 15) & KYBER_Q;
        r[3 * i + 0] = (uint8_t)(t0 >> 0);
        r[3 * i + 1] = (uint8_t)((t0 >> 8) | (t1 << 4));
        r[3 * i + 2] = (uint8_t)(t1 >> 4);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_tomsg(uint8_t msg[KYBER_INDCPA_MSGBYTES], const poly* a) {
    size_t i, j;
    uint16_t t;

    for (i = 0; i < KYBER_N / 8; i++) {
        msg[i] = 0;
        for (j = 0; j < 8; j++) {
            t = a->coeffs[8 * i + j];
            t += ((int16_t)t >> 15) & KYBER_Q;
            t = (((t << 1) + KYBER_Q / 2) / KYBER_Q) & 1;
            msg[i] |= t << j;
        }
    }
}

__device__ static void KeccakF1600_StatePermute(uint64_t* state) {
    int round;

    uint64_t Aba, Abe, Abi, Abo, Abu;
    uint64_t Aga, Age, Agi, Ago, Agu;
    uint64_t Aka, Ake, Aki, Ako, Aku;
    uint64_t Ama, Ame, Ami, Amo, Amu;
    uint64_t Asa, Ase, Asi, Aso, Asu;
    uint64_t BCa, BCe, BCi, BCo, BCu;
    uint64_t Da, De, Di, Do, Du;
    uint64_t Eba, Ebe, Ebi, Ebo, Ebu;
    uint64_t Ega, Ege, Egi, Ego, Egu;
    uint64_t Eka, Eke, Eki, Eko, Eku;
    uint64_t Ema, Eme, Emi, Emo, Emu;
    uint64_t Esa, Ese, Esi, Eso, Esu;

    // copyFromState(A, state)
    Aba = state[0];
    Abe = state[1];
    Abi = state[2];
    Abo = state[3];
    Abu = state[4];
    Aga = state[5];
    Age = state[6];
    Agi = state[7];
    Ago = state[8];
    Agu = state[9];
    Aka = state[10];
    Ake = state[11];
    Aki = state[12];
    Ako = state[13];
    Aku = state[14];
    Ama = state[15];
    Ame = state[16];
    Ami = state[17];
    Amo = state[18];
    Amu = state[19];
    Asa = state[20];
    Ase = state[21];
    Asi = state[22];
    Aso = state[23];
    Asu = state[24];

    for (round = 0; round < NROUNDS; round += 2) {
        //    prepareTheta
        BCa = Aba ^ Aga ^ Aka ^ Ama ^ Asa;
        BCe = Abe ^ Age ^ Ake ^ Ame ^ Ase;
        BCi = Abi ^ Agi ^ Aki ^ Ami ^ Asi;
        BCo = Abo ^ Ago ^ Ako ^ Amo ^ Aso;
        BCu = Abu ^ Agu ^ Aku ^ Amu ^ Asu;

        // thetaRhoPiChiIotaPrepareTheta(round  , A, E)
        Da = BCu ^ ROL(BCe, 1);
        De = BCa ^ ROL(BCi, 1);
        Di = BCe ^ ROL(BCo, 1);
        Do = BCi ^ ROL(BCu, 1);
        Du = BCo ^ ROL(BCa, 1);

        Aba ^= Da;
        BCa = Aba;
        Age ^= De;
        BCe = ROL(Age, 44);
        Aki ^= Di;
        BCi = ROL(Aki, 43);
        Amo ^= Do;
        BCo = ROL(Amo, 21);
        Asu ^= Du;
        BCu = ROL(Asu, 14);
        Eba = BCa ^ ((~BCe) & BCi);
        Eba ^= KeccakF_RoundConstants[round];
        Ebe = BCe ^ ((~BCi) & BCo);
        Ebi = BCi ^ ((~BCo) & BCu);
        Ebo = BCo ^ ((~BCu) & BCa);
        Ebu = BCu ^ ((~BCa) & BCe);

        Abo ^= Do;
        BCa = ROL(Abo, 28);
        Agu ^= Du;
        BCe = ROL(Agu, 20);
        Aka ^= Da;
        BCi = ROL(Aka, 3);
        Ame ^= De;
        BCo = ROL(Ame, 45);
        Asi ^= Di;
        BCu = ROL(Asi, 61);
        Ega = BCa ^ ((~BCe) & BCi);
        Ege = BCe ^ ((~BCi) & BCo);
        Egi = BCi ^ ((~BCo) & BCu);
        Ego = BCo ^ ((~BCu) & BCa);
        Egu = BCu ^ ((~BCa) & BCe);

        Abe ^= De;
        BCa = ROL(Abe, 1);
        Agi ^= Di;
        BCe = ROL(Agi, 6);
        Ako ^= Do;
        BCi = ROL(Ako, 25);
        Amu ^= Du;
        BCo = ROL(Amu, 8);
        Asa ^= Da;
        BCu = ROL(Asa, 18);
        Eka = BCa ^ ((~BCe) & BCi);
        Eke = BCe ^ ((~BCi) & BCo);
        Eki = BCi ^ ((~BCo) & BCu);
        Eko = BCo ^ ((~BCu) & BCa);
        Eku = BCu ^ ((~BCa) & BCe);

        Abu ^= Du;
        BCa = ROL(Abu, 27);
        Aga ^= Da;
        BCe = ROL(Aga, 36);
        Ake ^= De;
        BCi = ROL(Ake, 10);
        Ami ^= Di;
        BCo = ROL(Ami, 15);
        Aso ^= Do;
        BCu = ROL(Aso, 56);
        Ema = BCa ^ ((~BCe) & BCi);
        Eme = BCe ^ ((~BCi) & BCo);
        Emi = BCi ^ ((~BCo) & BCu);
        Emo = BCo ^ ((~BCu) & BCa);
        Emu = BCu ^ ((~BCa) & BCe);

        Abi ^= Di;
        BCa = ROL(Abi, 62);
        Ago ^= Do;
        BCe = ROL(Ago, 55);
        Aku ^= Du;
        BCi = ROL(Aku, 39);
        Ama ^= Da;
        BCo = ROL(Ama, 41);
        Ase ^= De;
        BCu = ROL(Ase, 2);
        Esa = BCa ^ ((~BCe) & BCi);
        Ese = BCe ^ ((~BCi) & BCo);
        Esi = BCi ^ ((~BCo) & BCu);
        Eso = BCo ^ ((~BCu) & BCa);
        Esu = BCu ^ ((~BCa) & BCe);

        //    prepareTheta
        BCa = Eba ^ Ega ^ Eka ^ Ema ^ Esa;
        BCe = Ebe ^ Ege ^ Eke ^ Eme ^ Ese;
        BCi = Ebi ^ Egi ^ Eki ^ Emi ^ Esi;
        BCo = Ebo ^ Ego ^ Eko ^ Emo ^ Eso;
        BCu = Ebu ^ Egu ^ Eku ^ Emu ^ Esu;

        // thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
        Da = BCu ^ ROL(BCe, 1);
        De = BCa ^ ROL(BCi, 1);
        Di = BCe ^ ROL(BCo, 1);
        Do = BCi ^ ROL(BCu, 1);
        Du = BCo ^ ROL(BCa, 1);

        Eba ^= Da;
        BCa = Eba;
        Ege ^= De;
        BCe = ROL(Ege, 44);
        Eki ^= Di;
        BCi = ROL(Eki, 43);
        Emo ^= Do;
        BCo = ROL(Emo, 21);
        Esu ^= Du;
        BCu = ROL(Esu, 14);
        Aba = BCa ^ ((~BCe) & BCi);
        Aba ^= KeccakF_RoundConstants[round + 1];
        Abe = BCe ^ ((~BCi) & BCo);
        Abi = BCi ^ ((~BCo) & BCu);
        Abo = BCo ^ ((~BCu) & BCa);
        Abu = BCu ^ ((~BCa) & BCe);

        Ebo ^= Do;
        BCa = ROL(Ebo, 28);
        Egu ^= Du;
        BCe = ROL(Egu, 20);
        Eka ^= Da;
        BCi = ROL(Eka, 3);
        Eme ^= De;
        BCo = ROL(Eme, 45);
        Esi ^= Di;
        BCu = ROL(Esi, 61);
        Aga = BCa ^ ((~BCe) & BCi);
        Age = BCe ^ ((~BCi) & BCo);
        Agi = BCi ^ ((~BCo) & BCu);
        Ago = BCo ^ ((~BCu) & BCa);
        Agu = BCu ^ ((~BCa) & BCe);

        Ebe ^= De;
        BCa = ROL(Ebe, 1);
        Egi ^= Di;
        BCe = ROL(Egi, 6);
        Eko ^= Do;
        BCi = ROL(Eko, 25);
        Emu ^= Du;
        BCo = ROL(Emu, 8);
        Esa ^= Da;
        BCu = ROL(Esa, 18);
        Aka = BCa ^ ((~BCe) & BCi);
        Ake = BCe ^ ((~BCi) & BCo);
        Aki = BCi ^ ((~BCo) & BCu);
        Ako = BCo ^ ((~BCu) & BCa);
        Aku = BCu ^ ((~BCa) & BCe);

        Ebu ^= Du;
        BCa = ROL(Ebu, 27);
        Ega ^= Da;
        BCe = ROL(Ega, 36);
        Eke ^= De;
        BCi = ROL(Eke, 10);
        Emi ^= Di;
        BCo = ROL(Emi, 15);
        Eso ^= Do;
        BCu = ROL(Eso, 56);
        Ama = BCa ^ ((~BCe) & BCi);
        Ame = BCe ^ ((~BCi) & BCo);
        Ami = BCi ^ ((~BCo) & BCu);
        Amo = BCo ^ ((~BCu) & BCa);
        Amu = BCu ^ ((~BCa) & BCe);

        Ebi ^= Di;
        BCa = ROL(Ebi, 62);
        Ego ^= Do;
        BCe = ROL(Ego, 55);
        Eku ^= Du;
        BCi = ROL(Eku, 39);
        Ema ^= Da;
        BCo = ROL(Ema, 41);
        Ese ^= De;
        BCu = ROL(Ese, 2);
        Asa = BCa ^ ((~BCe) & BCi);
        Ase = BCe ^ ((~BCi) & BCo);
        Asi = BCi ^ ((~BCo) & BCu);
        Aso = BCo ^ ((~BCu) & BCa);
        Asu = BCu ^ ((~BCa) & BCe);
    }

    // copyToState(state, A)
    state[0] = Aba;
    state[1] = Abe;
    state[2] = Abi;
    state[3] = Abo;
    state[4] = Abu;
    state[5] = Aga;
    state[6] = Age;
    state[7] = Agi;
    state[8] = Ago;
    state[9] = Agu;
    state[10] = Aka;
    state[11] = Ake;
    state[12] = Aki;
    state[13] = Ako;
    state[14] = Aku;
    state[15] = Ama;
    state[16] = Ame;
    state[17] = Ami;
    state[18] = Amo;
    state[19] = Amu;
    state[20] = Asa;
    state[21] = Ase;
    state[22] = Asi;
    state[23] = Aso;
    state[24] = Asu;
}
__device__ static void keccak_absorb(uint64_t* s, uint32_t r, const uint8_t* m, size_t mlen, uint8_t p) {
    size_t i;
    uint8_t t[200];

    /* Zero state */
    for (i = 0; i < 25; ++i) {
        s[i] = 0;
    }

    while (mlen >= r) {
        for (i = 0; i < r / 8; ++i) {
            s[i] ^= load64(m + 8 * i);
        }

        KeccakF1600_StatePermute(s);
        mlen -= r;
        m += r;
    }

    for (i = 0; i < r; ++i) {
        t[i] = 0;
    }
    for (i = 0; i < mlen; ++i) {
        t[i] = m[i];
    }
    t[i] = p;
    t[r - 1] |= 128;
    for (i = 0; i < r / 8; ++i) {
        s[i] ^= load64(t + 8 * i);
    }
}
__device__ void shake128_ctx_release(shake128ctx* state)
{
    free(state->ctx);
}
__device__ void shake128_absorb(shake128ctx* state, const uint8_t* input, size_t inlen)
{
    state->ctx = (uint64_t*)malloc(PQC_SHAKECTX_BYTES);

    keccak_absorb(state->ctx, SHAKE128_RATE, input, inlen, 0x1F);
}
__device__ static void keccak_squeezeblocks(uint8_t* h, size_t nblocks, uint64_t* s, uint32_t r) {
    while (nblocks > 0) {
        KeccakF1600_StatePermute(s);
        for (size_t i = 0; i < (r >> 3); i++) {
            store64(h + 8 * i, s[i]);
        }
        h += r;
        nblocks--;
    }
}
__device__ void shake128_squeezeblocks(uint8_t* output, size_t nblocks, shake128ctx* state)
{
    keccak_squeezeblocks(output, nblocks, state->ctx, SHAKE128_RATE);
}

__device__ void PQCLEAN_KYBER512_CLEAN_poly_compress(uint8_t r[KYBER_POLYCOMPRESSEDBYTES], const poly* a) {
    size_t i, j;
    int16_t u;
    uint8_t t[8];

    for (i = 0; i < KYBER_N / 8; i++) {
        for (j = 0; j < 8; j++) {
            // map to positive standard representatives
            u = a->coeffs[8 * i + j];
            u += (u >> 15) & KYBER_Q;
            t[j] = ((((uint16_t)u << 4) + KYBER_Q / 2) / KYBER_Q) & 15;
        }

        r[0] = t[0] | (t[1] << 4);
        r[1] = t[2] | (t[3] << 4);
        r[2] = t[4] | (t[5] << 4);
        r[3] = t[6] | (t[7] << 4);
        r += 4;
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_compress(uint8_t r[KYBER_POLYVECCOMPRESSEDBYTES], const polyvec* a) {
    unsigned int i, j, k;

    uint16_t t[4];
    for (i = 0; i < KYBER_K; i++) {
        for (j = 0; j < KYBER_N / 4; j++) {
            for (k = 0; k < 4; k++) {
                t[k] = a->vec[i].coeffs[4 * j + k];
                t[k] += ((int16_t)t[k] >> 15) & KYBER_Q;
                t[k] = ((((uint32_t)t[k] << 10) + KYBER_Q / 2) / KYBER_Q) & 0x3ff;
            }

            r[0] = (uint8_t)(t[0] >> 0);
            r[1] = (uint8_t)((t[0] >> 8) | (t[1] << 2));
            r[2] = (uint8_t)((t[1] >> 6) | (t[2] << 4));
            r[3] = (uint8_t)((t[2] >> 4) | (t[3] << 6));
            r[4] = (uint8_t)(t[3] >> 2);
            r += 5;
        }
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_tobytes(uint8_t r[KYBER_POLYVECBYTES], const polyvec* a)
{
    unsigned int i;
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_tobytes(r + i * KYBER_POLYBYTES, &a->vec[i]);
    }
}
__device__ static void pack_sk(uint8_t r[KYBER_INDCPA_SECRETKEYBYTES], polyvec* sk)
{
    PQCLEAN_KYBER512_CLEAN_polyvec_tobytes(r, sk);
}
__device__ static void pack_pk(uint8_t r[KYBER_INDCPA_PUBLICKEYBYTES], polyvec* pk, const uint8_t seed[KYBER_SYMBYTES])
{
    size_t i;
    PQCLEAN_KYBER512_CLEAN_polyvec_tobytes(r, pk);
    for (i = 0; i < KYBER_SYMBYTES; i++) {
        r[i + KYBER_POLYVECBYTES] = seed[i];
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_kyber_shake128_absorb(xof_state* state, const uint8_t seed[KYBER_SYMBYTES], uint8_t x, uint8_t y)
{
    uint8_t extseed[KYBER_SYMBYTES + 2];

    memcpy(extseed, seed, KYBER_SYMBYTES);
    extseed[KYBER_SYMBYTES + 0] = x;
    extseed[KYBER_SYMBYTES + 1] = y;

    shake128_absorb(state, extseed, sizeof(extseed));
}
__device__ static unsigned int rej_uniform(int16_t* r, unsigned int len, const uint8_t* buf, unsigned int buflen)
{
    unsigned int ctr, pos;
    uint16_t val0, val1;

    ctr = pos = 0;
    while (ctr < len && pos + 3 <= buflen) {
        val0 = ((buf[pos + 0] >> 0) | ((uint16_t)buf[pos + 1] << 8)) & 0xFFF;
        val1 = ((buf[pos + 1] >> 4) | ((uint16_t)buf[pos + 2] << 4)) & 0xFFF;
        pos += 3;

        if (val0 < KYBER_Q) {
            r[ctr++] = val0;
        }
        if (ctr < len && val1 < KYBER_Q) {
            r[ctr++] = val1;
        }
    }

    return ctr;
}
__device__ void PQCLEAN_KYBER512_CLEAN_gen_matrix(polyvec* a, const uint8_t seed[KYBER_SYMBYTES], int transposed) {
    unsigned int ctr, i, j, k;
    unsigned int buflen, off;
    uint8_t buf[GEN_MATRIX_NBLOCKS * XOF_BLOCKBYTES + 2];
    xof_state state;

    for (i = 0; i < KYBER_K; i++) {
        for (j = 0; j < KYBER_K; j++) {
            if (transposed) {
                xof_absorb(&state, seed, (uint8_t)i, (uint8_t)j);
            }
            else {
                xof_absorb(&state, seed, (uint8_t)j, (uint8_t)i);
            }

            xof_squeezeblocks(buf, GEN_MATRIX_NBLOCKS, &state);
            buflen = GEN_MATRIX_NBLOCKS * XOF_BLOCKBYTES;
            ctr = rej_uniform(a[i].vec[j].coeffs, KYBER_N, buf, buflen);

            while (ctr < KYBER_N) {
                off = buflen % 3;
                for (k = 0; k < off; k++) {
                    buf[k] = buf[buflen - off + k];
                }
                xof_squeezeblocks(buf + off, 1, &state);
                buflen = off + XOF_BLOCKBYTES;
                ctr += rej_uniform(a[i].vec[j].coeffs + ctr, KYBER_N - ctr, buf, buflen);
            }
            xof_ctx_release(&state);
        }
    }
}

__device__ void sha3_256(uint8_t* output, const uint8_t* input, size_t inlen)
{
    uint64_t s[25];
    uint8_t t[SHA3_256_RATE];

    /* Absorb input */
    keccak_absorb(s, SHA3_256_RATE, input, inlen, 0x06);

    /* Squeeze output */
    keccak_squeezeblocks(t, 1, s, SHA3_256_RATE);

    for (size_t i = 0; i < 32; i++) {
        output[i] = t[i];
    }
}
__device__ void sha3_512(uint8_t* output, const uint8_t* input, size_t inlen) {
    uint64_t s[25];
    uint8_t t[SHA3_512_RATE];

    /* Absorb input */
    keccak_absorb(s, SHA3_512_RATE, input, inlen, 0x06);

    /* Squeeze output */
    keccak_squeezeblocks(t, 1, s, SHA3_512_RATE);

    for (size_t i = 0; i < 64; i++) {
        output[i] = t[i];
    }
}
__device__ void shake256_absorb(shake256ctx* state, const uint8_t* input, size_t inlen)
{
    state->ctx = (uint64_t*)malloc(PQC_SHAKECTX_BYTES);

    keccak_absorb(state->ctx, SHAKE256_RATE, input, inlen, 0x1F);
}
__device__ void shake256_squeezeblocks(uint8_t* output, size_t nblocks, shake256ctx* state)
{
    keccak_squeezeblocks(output, nblocks, state->ctx, SHAKE256_RATE);
}
__device__ void shake256_ctx_release(shake256ctx* state)
{
    free(state->ctx);
}
__device__ void shake256(uint8_t* output, size_t outlen, const uint8_t* input, size_t inlen)
{
    size_t nblocks = outlen / SHAKE256_RATE;
    uint8_t t[SHAKE256_RATE];
    shake256ctx s;

    shake256_absorb(&s, input, inlen);
    shake256_squeezeblocks(output, nblocks, &s);

    output += nblocks * SHAKE256_RATE;
    outlen -= nblocks * SHAKE256_RATE;

    if (outlen) {
        shake256_squeezeblocks(t, 1, &s);
        for (size_t i = 0; i < outlen; ++i) {
            output[i] = t[i];
        }
    }
    shake256_ctx_release(&s);
}

__device__ static void cbd3(poly* r, const uint8_t buf[3 * KYBER_N / 4])
{
    unsigned int i, j;
    uint32_t t, d;
    int16_t a, b;

    for (i = 0; i < KYBER_N / 4; i++) {
        t = load24_littleendian(buf + 3 * i);
        d = t & 0x00249249;
        d += (t >> 1) & 0x00249249;
        d += (t >> 2) & 0x00249249;

        for (j = 0; j < 4; j++) {
            a = (d >> (6 * j + 0)) & 0x7;
            b = (d >> (6 * j + 3)) & 0x7;
            r->coeffs[4 * i + j] = a - b;
        }
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_cbd_eta1(poly* r, const uint8_t buf[KYBER_ETA1 * KYBER_N / 4])
{
    cbd3(r, buf);
}
__device__ void PQCLEAN_KYBER512_CLEAN_kyber_shake256_prf(uint8_t* out, size_t outlen, const uint8_t key[KYBER_SYMBYTES], uint8_t nonce) {
    uint8_t extkey[KYBER_SYMBYTES + 1];

    memcpy(extkey, key, KYBER_SYMBYTES);
    extkey[KYBER_SYMBYTES] = nonce;

    shake256(out, outlen, extkey, sizeof(extkey));
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta1(poly* r, const uint8_t seed[KYBER_SYMBYTES], uint8_t nonce) {
    uint8_t buf[KYBER_ETA1 * KYBER_N / 4];
    prf(buf, sizeof(buf), seed, nonce);
    PQCLEAN_KYBER512_CLEAN_poly_cbd_eta1(r, buf);
}
__device__ int16_t PQCLEAN_KYBER512_CLEAN_montgomery_reduce(int32_t a)
{
    int16_t t;

    t = (int16_t)a * QINV;
    t = (a - (int32_t)t * KYBER_Q) >> 16;
    return t;
}
__device__ static int16_t fqmul(int16_t a, int16_t b)
{
    return PQCLEAN_KYBER512_CLEAN_montgomery_reduce((int32_t)a * b);
}
__device__ void PQCLEAN_KYBER512_CLEAN_ntt(int16_t r[256])
{
    unsigned int len, start, j, k;
    int16_t t, zeta;

    k = 1;
    for (len = 128; len >= 2; len >>= 1) {
        for (start = 0; start < 256; start = j + len) {
            zeta = PQCLEAN_KYBER512_CLEAN_zetas[k++];
            for (j = start; j < start + len; j++) {
                t = fqmul(zeta, r[j + len]);
                r[j + len] = r[j] - t;
                r[j] = r[j] + t;
            }
        }
    }
}
__device__ int16_t PQCLEAN_KYBER512_CLEAN_barrett_reduce(int16_t a) {
    int16_t t;
    const int16_t v = ((1 << 26) + KYBER_Q / 2) / KYBER_Q;

    t = ((int32_t)v * a + (1 << 25)) >> 26;
    t *= KYBER_Q;
    return a - t;
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_sub(poly* r, const poly* a, const poly* b) {
    size_t i;
    for (i = 0; i < KYBER_N; i++) {
        r->coeffs[i] = a->coeffs[i] - b->coeffs[i];
    }
}

__device__ void PQCLEAN_KYBER512_CLEAN_poly_reduce(poly* r)
{
    size_t i;
    for (i = 0; i < KYBER_N; i++) {
        r->coeffs[i] = PQCLEAN_KYBER512_CLEAN_barrett_reduce(r->coeffs[i]);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_ntt(poly* r)
{
    PQCLEAN_KYBER512_CLEAN_ntt(r->coeffs);
    PQCLEAN_KYBER512_CLEAN_poly_reduce(r);
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_ntt(polyvec* r)
{
    unsigned int i;
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_ntt(&r->vec[i]);
    }
}

__device__ void PQCLEAN_KYBER512_CLEAN_basemul(int16_t r[2], const int16_t a[2], const int16_t b[2], int16_t zeta)
{
    r[0] = fqmul(a[1], b[1]);
    r[0] = fqmul(r[0], zeta);
    r[0] += fqmul(a[0], b[0]);
    r[1] = fqmul(a[0], b[1]);
    r[1] += fqmul(a[1], b[0]);
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_basemul_montgomery(poly* r, const poly* a, const poly* b)
{
    size_t i;
    for (i = 0; i < KYBER_N / 4; i++) {
        PQCLEAN_KYBER512_CLEAN_basemul(&r->coeffs[4 * i], &a->coeffs[4 * i], &b->coeffs[4 * i], PQCLEAN_KYBER512_CLEAN_zetas[64 + i]);
        PQCLEAN_KYBER512_CLEAN_basemul(&r->coeffs[4 * i + 2], &a->coeffs[4 * i + 2], &b->coeffs[4 * i + 2], -PQCLEAN_KYBER512_CLEAN_zetas[64 + i]);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_add(poly* r, const poly* a, const poly* b)
{
    size_t i;
    for (i = 0; i < KYBER_N; i++) {
        r->coeffs[i] = a->coeffs[i] + b->coeffs[i];
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_basemul_acc_montgomery(poly* r, const polyvec* a, const polyvec* b)
{
    unsigned int i;
    poly t;

    PQCLEAN_KYBER512_CLEAN_poly_basemul_montgomery(r, &a->vec[0], &b->vec[0]);
    for (i = 1; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_basemul_montgomery(&t, &a->vec[i], &b->vec[i]);
        PQCLEAN_KYBER512_CLEAN_poly_add(r, r, &t);
    }

    PQCLEAN_KYBER512_CLEAN_poly_reduce(r);
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_tomont(poly* r)
{
    size_t i;
    const int16_t f = (1ULL << 32) % KYBER_Q;
    for (i = 0; i < KYBER_N; i++) {
        r->coeffs[i] = PQCLEAN_KYBER512_CLEAN_montgomery_reduce((int32_t)r->coeffs[i] * f);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_add(polyvec* r, const polyvec* a, const polyvec* b)
{
    unsigned int i;
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_add(&r->vec[i], &a->vec[i], &b->vec[i]);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_reduce(polyvec* r) {
    unsigned int i;
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_reduce(&r->vec[i]);
    }
}

__device__ void PQCLEAN_KYBER512_CLEAN_indcpa_keypair(uint8_t pk[KYBER_INDCPA_PUBLICKEYBYTES], uint8_t sk[KYBER_INDCPA_SECRETKEYBYTES])
{
    unsigned int i;
    uint8_t buf[2 * KYBER_SYMBYTES];                    // �ؽ�, �õ带 �����ϴ� �迭
    const uint8_t* publicseed = buf;                    // ����Ű�� seed���� ������ ���� publicseed ����
    const uint8_t* noiseseed = buf + KYBER_SYMBYTES;    // noise���� seed���� �����ϱ� ���� noiseseed ����

    /** ���� 3 **/
    uint8_t nonce = 0;

    polyvec a[KYBER_K], e, pkpv, skpv;

    /** ���� 1 **/
    //randombytes(buf, KYBER_SYMBYTES);                 // ������ 32����Ʈ ����
    for (int i = 0; i < 32; i++) {
        buf[i] = i;
    }

    /** ���� 2 **/
    hash_g(buf, buf, KYBER_SYMBYTES);                   // SHA3_512�� �������� Seed�� ������� -> SHA3_512�� �� 32byte�� ����Ű�� seed��, �� 32byte�� noise�� seed�� ���� ���

    /** ���� 4 ~ 8 **/
    gen_a(a, publicseed);                               //������� a ���� (2x2)

    /** ���� 9 ~ 12 **/
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta1(&skpv.vec[i], noiseseed, nonce++);    //skpv.vec[0], noiseseed, 0 -> skpv.vec[1], noiseseed, 1
    }   // ���Ű s ����

    /** ���� 13 ~ 16 **/
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta1(&e.vec[i], noiseseed, nonce++);       //e.vec[0], noiseseed, 2 -> e.vec[1], noiseseed, 3
    }   // �������׽� e ����


    /** ���� 17 ~ 18 **/
    PQCLEAN_KYBER512_CLEAN_polyvec_ntt(&skpv);  // s�� NTT��ȯ!
    PQCLEAN_KYBER512_CLEAN_polyvec_ntt(&e);     // e�� NTT ��ȯ!


    /** ���� 19 **/
    // matrix-vector multiplication
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_polyvec_basemul_acc_montgomery(&pkpv.vec[i], &a[i], &skpv);
        PQCLEAN_KYBER512_CLEAN_poly_tomont(&pkpv.vec[i]);
    } // X = A*s

    PQCLEAN_KYBER512_CLEAN_polyvec_add(&pkpv, &pkpv, &e);       // X + e
    PQCLEAN_KYBER512_CLEAN_polyvec_reduce(&pkpv);               // ���׽� ����鿡 ���� mod q

    /** ���� 20 ~ 21 **/
    pack_sk(sk, &skpv);                                         //sk ����ȭ -> �� ����Ʈ �迭�� ����
    pack_pk(pk, &pkpv, publicseed);                             //pk ����ȭ -> �� ����Ʈ �迭�� ����
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_frombytes(poly* r, const uint8_t a[KYBER_POLYBYTES]) {
    size_t i;
    for (i = 0; i < KYBER_N / 2; i++) {
        r->coeffs[2 * i] = ((a[3 * i + 0] >> 0) | ((uint16_t)a[3 * i + 1] << 8)) & 0xFFF;
        r->coeffs[2 * i + 1] = ((a[3 * i + 1] >> 4) | ((uint16_t)a[3 * i + 2] << 4)) & 0xFFF;
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_frombytes(polyvec* r, const uint8_t a[KYBER_POLYVECBYTES]) {
    unsigned int i;
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_frombytes(&r->vec[i], a + i * KYBER_POLYBYTES);
    }
}

__device__ static void unpack_pk(polyvec* pk, uint8_t seed[KYBER_SYMBYTES], const uint8_t packedpk[KYBER_INDCPA_PUBLICKEYBYTES])
{
    size_t i;
    /** ���� 2 **/
    PQCLEAN_KYBER512_CLEAN_polyvec_frombytes(pk, packedpk);
    for (i = 0; i < KYBER_SYMBYTES; i++) {
        /** ���� 3 **/
        seed[i] = packedpk[i + KYBER_POLYVECBYTES];
    }
}
__device__ static void cbd2(poly* r, const uint8_t buf[2 * KYBER_N / 4]) {
    unsigned int i, j;
    uint32_t t, d;
    int16_t a, b;

    for (i = 0; i < KYBER_N / 8; i++) {
        t = load32_littleendian(buf + 4 * i);
        d = t & 0x55555555;
        d += (t >> 1) & 0x55555555;

        for (j = 0; j < 8; j++) {
            a = (d >> (4 * j + 0)) & 0x3;
            b = (d >> (4 * j + 2)) & 0x3;
            r->coeffs[8 * i + j] = a - b;
        }
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_cbd_eta2(poly* r, const uint8_t buf[KYBER_ETA2 * KYBER_N / 4]) {
    cbd2(r, buf);
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta2(poly* r, const uint8_t seed[KYBER_SYMBYTES], uint8_t nonce) {
    uint8_t buf[KYBER_ETA2 * KYBER_N / 4];
    prf(buf, sizeof(buf), seed, nonce);
    PQCLEAN_KYBER512_CLEAN_poly_cbd_eta2(r, buf);
}

__device__ void PQCLEAN_KYBER512_CLEAN_invntt(int16_t r[256]) {
    unsigned int start, len, j, k;
    int16_t t, zeta;
    const int16_t f = 1441; // mont^2/128

    k = 127;
    for (len = 2; len <= 128; len <<= 1) {
        for (start = 0; start < 256; start = j + len) {
            zeta = PQCLEAN_KYBER512_CLEAN_zetas[k--];
            for (j = start; j < start + len; j++) {
                t = r[j];
                r[j] = PQCLEAN_KYBER512_CLEAN_barrett_reduce(t + r[j + len]);
                r[j + len] = r[j + len] - t;
                r[j + len] = fqmul(zeta, r[j + len]);
            }
        }
    }

    for (j = 0; j < 256; j++) {
        r[j] = fqmul(r[j], f);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_poly_invntt_tomont(poly* r) {
    PQCLEAN_KYBER512_CLEAN_invntt(r->coeffs);
}
__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_invntt_tomont(polyvec* r) {
    unsigned int i;
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_invntt_tomont(&r->vec[i]);
    }
}

__device__ static void pack_ciphertext(uint8_t r[KYBER_INDCPA_BYTES], polyvec* b, poly* v) {
    PQCLEAN_KYBER512_CLEAN_polyvec_compress(r, b);
    PQCLEAN_KYBER512_CLEAN_poly_compress(r + KYBER_POLYVECCOMPRESSEDBYTES, v);
}
__device__ void PQCLEAN_KYBER512_CLEAN_indcpa_enc(uint8_t* c, const uint8_t m[KYBER_INDCPA_MSGBYTES], const uint8_t pk[KYBER_INDCPA_PUBLICKEYBYTES], const uint8_t coins[KYBER_SYMBYTES])   //�˰��� 4
{
    unsigned int i;
    uint8_t seed[KYBER_SYMBYTES];
    uint8_t nonce = 0;                      // /** ���� 1 **/
    polyvec sp, pkpv, ep, at[KYBER_K], b;
    poly v, k, epp;

    /** ���� 2, 3 **/
    unpack_pk(&pkpv, seed, pk);                 //array -> module �̶�� �����ϴ°� ����


    /** ���� 4 ~ 8 **/
    gen_at(at, seed);

    /** ���� 9 ~ 12 **/
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta1(sp.vec + i, coins, nonce++);
    }

    /** ���� 13 ~ 16 **/
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta2(ep.vec + i, coins, nonce++);
    }

    /** ���� 17 **/
    PQCLEAN_KYBER512_CLEAN_poly_getnoise_eta2(&epp, coins, nonce++);

    /** ���� 18 **/
    PQCLEAN_KYBER512_CLEAN_polyvec_ntt(&sp);

    /** ���� 19 - mul(AT, r) �κ� **/
    // matrix-vector multiplication
    for (i = 0; i < KYBER_K; i++) {
        PQCLEAN_KYBER512_CLEAN_polyvec_basemul_acc_montgomery(&b.vec[i], &at[i], &sp);
    }

    /** ���� 20 - mul(tT, r) �κ� **/
    PQCLEAN_KYBER512_CLEAN_polyvec_basemul_acc_montgomery(&v, &pkpv, &sp);

    /** ���� 19 invNTT(mul(AT, r)) �κ� **/
    PQCLEAN_KYBER512_CLEAN_polyvec_invntt_tomont(&b);

    /** ���� 20 invNTT(mul(tT, r)) �κ� **/
    PQCLEAN_KYBER512_CLEAN_poly_invntt_tomont(&v);

    /** ���� 19 invNTT(mul(AT, r)) + e1 �κ� **/
    PQCLEAN_KYBER512_CLEAN_polyvec_add(&b, &b, &ep);

    /** ���� 20 invNTT(mul(tT, r)) + e2  + Decompress_q�κ� **/
    PQCLEAN_KYBER512_CLEAN_poly_add(&v, &v, &epp);  // + e2 ����
    PQCLEAN_KYBER512_CLEAN_poly_frommsg(&k, m);     //m�� ��(message)�� k ��(polynomial)�� ����, (Decompress_q(Decode_1(m), 1) ����)
    PQCLEAN_KYBER512_CLEAN_poly_add(&v, &v, &k);    // + k ����

    /** ���� 21 **/
    PQCLEAN_KYBER512_CLEAN_polyvec_reduce(&b);

    /** ���� 22 **/
    PQCLEAN_KYBER512_CLEAN_poly_reduce(&v);

    /** ���� 23 **/
    pack_ciphertext(c, &b, &v);
}

__device__ void PQCLEAN_KYBER512_CLEAN_crypto_kem_keypair(uint8_t* pk, uint8_t* sk)
{
    size_t i;
    PQCLEAN_KYBER512_CLEAN_indcpa_keypair(pk, sk);                      //�˰��� 3 -> KEY �����Լ� : KYBER�� CPA(Chosen Plaintext Attack) �������� �����ϴ� PKE ��Ŵ�� Ű���� ����
    for (i = 0; i < KYBER_INDCPA_PUBLICKEYBYTES; i++) {
        sk[i + KYBER_INDCPA_SECRETKEYBYTES] = pk[i];
    }
    hash_h(sk + KYBER_SECRETKEYBYTES - 2 * KYBER_SYMBYTES, pk, KYBER_PUBLICKEYBYTES);
    /* Value z for pseudo-random output on reject */
    randombytes_win32_randombytes(sk + KYBER_SECRETKEYBYTES - KYBER_SYMBYTES, KYBER_SYMBYTES);
}

__device__ void PQCLEAN_KYBER512_CLEAN_crypto_kem_enc(uint8_t* ct, uint8_t* ss, uint8_t* pk)  //�˰��� 7 Encapsulation
{
    uint8_t buf[2 * KYBER_SYMBYTES];
    /* Will contain key, coins */
    uint8_t kr[2 * KYBER_SYMBYTES];

    /** ���� 1 **/
    randombytes_win32_randombytes(buf, KYBER_SYMBYTES);

    /* Don't release system RNG output */
    /** ���� 2 **/
    hash_h(buf, buf, KYBER_SYMBYTES);

    /* Multitarget countermeasure for coins + contributory KEM */
    /** ���� 3 **/
    hash_h(buf + KYBER_SYMBYTES, pk, KYBER_PUBLICKEYBYTES);
    hash_g(kr, buf, 2 * KYBER_SYMBYTES);

    /* coins are in kr+KYBER_SYMBYTES */
    /** ���� 4 **/
    PQCLEAN_KYBER512_CLEAN_indcpa_enc(ct, buf, pk, kr + KYBER_SYMBYTES);

    /** ���� 5 **/
    /* overwrite coins in kr with H(c) */
    hash_h(kr + KYBER_SYMBYTES, ct, KYBER_CIPHERTEXTBYTES);
    /* hash concatenation of pre-k and H(c) to k */
    kdf(ss, kr, 2 * KYBER_SYMBYTES);
}

__device__ void PQCLEAN_KYBER512_CLEAN_polyvec_decompress(polyvec* r, const uint8_t a[KYBER_POLYVECCOMPRESSEDBYTES]) {
    unsigned int i, j, k;

    uint16_t t[4];
    for (i = 0; i < KYBER_K; i++) {
        for (j = 0; j < KYBER_N / 4; j++) {
            t[0] = (a[0] >> 0) | ((uint16_t)a[1] << 8);
            t[1] = (a[1] >> 2) | ((uint16_t)a[2] << 6);
            t[2] = (a[2] >> 4) | ((uint16_t)a[3] << 4);
            t[3] = (a[3] >> 6) | ((uint16_t)a[4] << 2);
            a += 5;

            for (k = 0; k < 4; k++) {
                r->vec[i].coeffs[4 * j + k] = ((uint32_t)(t[k] & 0x3FF) * KYBER_Q + 512) >> 10;
            }
        }
    }
}

__device__ void PQCLEAN_KYBER512_CLEAN_poly_decompress(poly* r, const uint8_t a[KYBER_POLYCOMPRESSEDBYTES]) {
    size_t i;

    for (i = 0; i < KYBER_N / 2; i++) {
        r->coeffs[2 * i + 0] = (((uint16_t)(a[0] & 15) * KYBER_Q) + 8) >> 4;
        r->coeffs[2 * i + 1] = (((uint16_t)(a[0] >> 4) * KYBER_Q) + 8) >> 4;
        a += 1;
    }
}

__device__ static void unpack_ciphertext(polyvec* b, poly* v, const uint8_t c[KYBER_INDCPA_BYTES]) {
    PQCLEAN_KYBER512_CLEAN_polyvec_decompress(b, c);
    PQCLEAN_KYBER512_CLEAN_poly_decompress(v, c + KYBER_POLYVECCOMPRESSEDBYTES);
}
__device__ static void unpack_sk(polyvec* sk, const uint8_t packedsk[KYBER_INDCPA_SECRETKEYBYTES]) {
    PQCLEAN_KYBER512_CLEAN_polyvec_frombytes(sk, packedsk);
}

__device__ int PQCLEAN_KYBER512_CLEAN_verify(const uint8_t* a, const uint8_t* b, size_t len) {
    size_t i;
    uint8_t r = 0;

    for (i = 0; i < len; i++) {
        r |= a[i] ^ b[i];
    }

    return ((~(uint64_t)r) + 1) >> 63;
}

__device__ void PQCLEAN_KYBER512_CLEAN_cmov(uint8_t* r, const uint8_t* x, size_t len, uint8_t b) {
    size_t i;

    b = -b;
    for (i = 0; i < len; i++) {
        r[i] ^= b & (r[i] ^ x[i]);
    }
}
__device__ void PQCLEAN_KYBER512_CLEAN_indcpa_dec(uint8_t m[KYBER_INDCPA_MSGBYTES], const uint8_t c[KYBER_INDCPA_BYTES], const uint8_t sk[KYBER_INDCPA_SECRETKEYBYTES])
{
    polyvec b, skpv;
    poly v, mp;

    unpack_ciphertext(&b, &v, c);
    unpack_sk(&skpv, sk);

    PQCLEAN_KYBER512_CLEAN_polyvec_ntt(&b);
    PQCLEAN_KYBER512_CLEAN_polyvec_basemul_acc_montgomery(&mp, &skpv, &b);
    PQCLEAN_KYBER512_CLEAN_poly_invntt_tomont(&mp);

    PQCLEAN_KYBER512_CLEAN_poly_sub(&mp, &v, &mp);
    PQCLEAN_KYBER512_CLEAN_poly_reduce(&mp);

    PQCLEAN_KYBER512_CLEAN_poly_tomsg(m, &mp);
}
__device__ void PQCLEAN_KYBER512_CLEAN_crypto_kem_dec(uint8_t* ss, const uint8_t* ct, const uint8_t* sk)
{
    size_t i;
    int fail;
    uint8_t buf[2 * KYBER_SYMBYTES];
    /* Will contain key, coins */
    uint8_t kr[2 * KYBER_SYMBYTES];
    uint8_t cmp[KYBER_CIPHERTEXTBYTES];
    const uint8_t* pk = sk + KYBER_INDCPA_SECRETKEYBYTES;

    PQCLEAN_KYBER512_CLEAN_indcpa_dec(buf, ct, sk);

    /* Multitarget countermeasure for coins + contributory KEM */
    for (i = 0; i < KYBER_SYMBYTES; i++) {
        buf[KYBER_SYMBYTES + i] = sk[KYBER_SECRETKEYBYTES - 2 * KYBER_SYMBYTES + i];
    }
    hash_g(kr, buf, 2 * KYBER_SYMBYTES);

    /* coins are in kr+KYBER_SYMBYTES */
    PQCLEAN_KYBER512_CLEAN_indcpa_enc(cmp, buf, pk, kr + KYBER_SYMBYTES);

    fail = PQCLEAN_KYBER512_CLEAN_verify(ct, cmp, KYBER_CIPHERTEXTBYTES);

    /* overwrite coins in kr with H(c) */
    hash_h(kr + KYBER_SYMBYTES, ct, KYBER_CIPHERTEXTBYTES);

    /* Overwrite pre-k with z on re-encryption failure */
    PQCLEAN_KYBER512_CLEAN_cmov(kr, sk + KYBER_SECRETKEYBYTES - KYBER_SYMBYTES, KYBER_SYMBYTES, (uint8_t)fail);

    /* hash concatenation of pre-k and H(c) to k */
    kdf(ss, kr, 2 * KYBER_SYMBYTES);
}

__global__ void GPU_Kyber(uint8_t* pk, uint8_t* sk, uint8_t* ct, uint8_t* ss, uint8_t* ss2)
{
    int tid;

    tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    PQCLEAN_KYBER512_CLEAN_crypto_kem_keypair(pk + PQCLEAN_KYBER512_CLEAN_CRYPTO_PUBLICKEYBYTES * tid, sk + PQCLEAN_KYBER512_CLEAN_CRYPTO_SECRETKEYBYTES * tid);			// KEY ���� + KEY ��ȯ
    PQCLEAN_KYBER512_CLEAN_crypto_kem_enc(ct + KYBER_CIPHERTEXTBYTES * tid, ss + 32 * tid, pk + PQCLEAN_KYBER512_CLEAN_CRYPTO_PUBLICKEYBYTES * tid);
    PQCLEAN_KYBER512_CLEAN_crypto_kem_dec(ss2 + 32 * tid, ct + KYBER_CIPHERTEXTBYTES * tid, sk + PQCLEAN_KYBER512_CLEAN_CRYPTO_SECRETKEYBYTES * tid);
}



void test_Kyber(uint64_t blocksize, uint64_t threadsize)
{
    uint8_t* pk = NULL;
    uint8_t* sk = NULL;
    uint8_t* ct = NULL;
    uint8_t* ss = NULL;
    uint8_t* ss2 = NULL;

    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;

    pk = (uint8_t*)malloc(PQCLEAN_KYBER512_CLEAN_CRYPTO_PUBLICKEYBYTES * blocksize * threadsize); //PQCLEAN_KYBER512_CLEAN_CRYPTO_PUBLICKEYBYTES -> (2 * 384 + 32) * blocksize * threadsize
    sk = (uint8_t*)malloc(PQCLEAN_KYBER512_CLEAN_CRYPTO_SECRETKEYBYTES * blocksize * threadsize); //PQCLEAN_KYBER512_CLEAN_CRYPTO_SECRETKEYBYTES -> ((2 * 384) + (2 * 384 + 32) + (2 * 32)) * blocksize * threadsize
    ct = (uint8_t*)malloc(KYBER_CIPHERTEXTBYTES * blocksize * threadsize);
    ss = (uint8_t*)malloc(32 * blocksize * threadsize);
    ss2 = (uint8_t*)malloc(32 * blocksize * threadsize);

    uint8_t* GPU_pk;
    uint8_t* GPU_sk;
    uint8_t* GPU_ct;
    uint8_t* GPU_ss;
    uint8_t* GPU_ss2;

    hipMalloc((void**)&GPU_pk, PQCLEAN_KYBER512_CLEAN_CRYPTO_PUBLICKEYBYTES * blocksize * threadsize);
    hipMalloc((void**)&GPU_sk, PQCLEAN_KYBER512_CLEAN_CRYPTO_SECRETKEYBYTES * blocksize * threadsize);
    hipMalloc((void**)&GPU_ct, KYBER_CIPHERTEXTBYTES * blocksize * threadsize);
    hipMalloc((void**)&GPU_ss, 32 * blocksize * threadsize);
    hipMalloc((void**)&GPU_ss2, 32 * blocksize * threadsize);


    printf("\nStart...\n");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < N_TIMES; i++)
        GPU_Kyber << <blocksize, threadsize >> > (GPU_pk, GPU_sk, GPU_ct, GPU_ss, GPU_ss2);

    printf("%d\n", N_TIMES);

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    elapsed_time_ms /= N_TIMES;

    printf("elapsed time: %4.2f ms\n\n", elapsed_time_ms);
    
    elapsed_time_ms = (1000 / elapsed_time_ms) * blocksize * threadsize;
    printf("Grid : %ld, Block : %ld, Performance : %4.2f Kyber/s\n", blocksize, threadsize, elapsed_time_ms);

    hipMemcpy(ss, GPU_ss, 32 * blocksize * threadsize, hipMemcpyDeviceToHost);
    hipMemcpy(ss2, GPU_ss2, 32 * blocksize * threadsize, hipMemcpyDeviceToHost);

    if (!memcmp(ss, ss2, 32 * blocksize * threadsize))
        printf("\n\nSuccess!\n\n");
    else
        printf("\n\nFail\n\n");

    getchar();
    getchar();

    printf("ss_0: = \n");
    for (int i = 0; i < 32 * blocksize * threadsize; i++) {
        printf("%02X ", ss[i]);

        if ((i + 1) % 32 == 0)
        {
            printf("\nss_%d: = \n", (i + 1)/32);
            printf("\n");
        }
    }
    printf("\n\n");

    printf("ss2_0: = \n");
    for (int i = 0; i < 32 * blocksize * threadsize; i++) {
        printf("%02X ", ss2[i]);

        if ((i + 1) % 32 == 0)
        {
            printf("\nss2_%d: = \n", (i + 1) / 32);
            printf("\n");
        }
    }

    hipFree(GPU_pk);
    hipFree(GPU_sk);
    hipFree(GPU_ct);
    hipFree(GPU_ss);
    hipFree(GPU_ss2);
    free(pk);
    free(sk);
    free(ct);
    free(ss);
    free(ss2);
}

int main()
{
    test_Kyber(256, 256);

    return 0;
}